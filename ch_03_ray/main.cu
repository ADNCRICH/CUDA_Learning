#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../ch_02_vector/vec3.h"
#include "ray.h"
#include "stb_image_write.h"
using namespace std;

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

template <typename com_t>
__device__ vec3<com_t> color(ray<com_t> r) {
    float t = (unit_vector(r.direction()).y() + 1) * 0.5;                              // (-inf, inf) -> (0, 1)
    return ((1 - t) * vec3<com_t>(1, 1, 1) + t * vec3<com_t>(0.5, 0.7, 1)) * 255.99f;  // blend color white and blue
}

template <typename out_t, typename com_t>
__global__ void color(vec3<out_t> *output, int X, int Y, vec3<com_t> lower_left_corner, vec3<com_t> horizontal, vec3<com_t> vertical, vec3<com_t> origin) {
    int x = blockDim.x * blockIdx.x + threadIdx.x, y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= X || y >= Y) return;
    com_t u = float(x) / float(X), v = float(Y - y - 1) / float(Y);
    ray<float> r(origin, lower_left_corner + u * horizontal + v * vertical);
    output[X * y + x] = color(r);
}

int main() {
    int nx = 2000, ny = 1000;
    int thread_size = 16;
    vec3<uint8_t> *output;
    hipMallocManaged(&output, nx * ny * sizeof(vec3<float>));

    dim3 threads(thread_size, thread_size);
    dim3 blocks((nx + thread_size - 1) / thread_size, (ny + thread_size - 1) / thread_size);

    vec3<float> lower_left_corner(-2, -1, -1);
    vec3<float> horizontal(4, 0, 0);
    vec3<float> vertical(0, 2, 0);
    vec3<float> origin(0, 0, 0);

    color<<<blocks, threads>>>(output, nx, ny, lower_left_corner, horizontal, vertical, origin);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stbi_write_jpg("./background.jpg", nx, ny, 3, output, 100);

    checkCudaErrors(hipFree(output));
}